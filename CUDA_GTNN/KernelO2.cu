#include "hip/hip_runtime.h"
#include "KernelO2.cuh"
#include <hip/hip_runtime.h>

__global__ void simulate_step(
    const float* Q,
    float* vp,
    float* vn,
    int* spikesP,
    int* spikesN,
    int n,
    float dt,
    float tau,
    float vmax,
    float vth,
    float Lambda,
    float C
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    // Shared memory for voltage differences
    extern __shared__ float shared_mem[];
    float* shared_diff = shared_mem;  // n elements

    // Cache voltage differences in shared memory with coalesced access
    for (int j = threadIdx.x; j < n; j += blockDim.x) {
        shared_diff[j] = vp[j] - vn[j];
    }
    __syncthreads();

    float vpi = vp[i];
    float vni = vn[i];
    bool spikedP = false;
    bool spikedN = false;

    // Compute Q*(vp-vn) directly without tiling for better accuracy
    float deltaV = 0.0f;
    #pragma unroll 4
    for (int j = 0; j < n; j++) {
        deltaV += Q[i * n + j] * shared_diff[j];
    }
    __syncthreads();

    // Basic DC input (only neuron 0) - matching naive and O1 implementations
    float netI = (i == 0) ? 0.09f : 0.0f;

    // Gradient terms
    float Gp = vpi - netI + deltaV;
    float Gn = vni + netI - deltaV;

    // Check for spikes and update with refractory safeguard
    if (vpi > vth) { 
        Gp += C; 
        vpi = vth;
        spikedP = true;
    }
    if (vni > vth) { 
        Gn += C; 
        vni = vth;
        spikedN = true;
    }

    // Prevent denominator from getting too close to zero
    float eps = 1e-6f;
    float denomP = -vpi * Gp + Lambda * vmax;
    float denomN = -vni * Gn + Lambda * vmax;
    
    // Add small epsilon to prevent division by zero
    denomP = (fabsf(denomP) < eps) ? (denomP < 0 ? -eps : eps) : denomP;
    denomN = (fabsf(denomN) < eps) ? (denomN < 0 ? -eps : eps) : denomN;
    
    float dtTau = dt / tau;
    
    float numP = vpi * vpi - vmax * vmax;
    float numN = vni * vni - vmax * vmax;
    
    float dvp = dtTau * ((numP * Gp) / denomP);
    float dvn = dtTau * ((numN * Gn) / denomN);
    
    // Update voltages with bounds checking
    vp[i] = fmaxf(fminf(vpi + dvp, vmax), -vmax);
    vn[i] = fmaxf(fminf(vni + dvn, vmax), -vmax);

    // Atomically increment spike counters if needed
    if (spikedP) atomicAdd(spikesP, 1);
    if (spikedN) atomicAdd(spikesN, 1);
}